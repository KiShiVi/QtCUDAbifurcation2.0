#include "hip/hip_runtime.h"
﻿#include <stdexcept>
#include <iostream>
#include <fstream>
#include <iomanip>

#include "bifurcationKernel.cuh"

__host__ void bifurcation1D(
	double					in_tMax,
	int					in_nPts,
	double				in_h,
	double* in_initialConditions,
	double				in_paramValues1,
	double				in_paramValues2,
	int					in_nValue,
	double				in_prePeakFinderSliceK,
	int					in_thresholdValueOfMaxSignalValue,
	int					in_amountOfParams,
	int					in_discreteModelMode,
	int					in_prescaller,
	double* in_params,
	int					in_mode,
	double				in_memoryLimit,
	std::string			in_outPath,
	bool				in_debug,
	std::atomic<int>& progress)
{
	size_t amountOfTPoints = in_tMax / in_h / in_prescaller;

	double* globalParamValues = nullptr;
	globalParamValues = (double*)malloc(sizeof(double) * in_nPts);
	linspace(in_paramValues1, in_paramValues2, in_nPts, globalParamValues);

	size_t freeMemory;
	size_t totalMemory;

	hipMemGetInfo(&freeMemory, &totalMemory);

	freeMemory *= in_memoryLimit * 0.95;

	double maxMemoryLimit = sizeof(double) * (amountOfTPoints + 2 + in_amountOfParams) + sizeof(int);

	size_t nPtsLimiter = freeMemory / maxMemoryLimit;
	//ne takaya yzh huita ebanaya
	if (nPtsLimiter <= 0)
	{
		if (in_debug)
			std::cout << "\nVery low memory size. Increase the MEMORY_LIMIT!" << "\n";
		exit(1);
	}

	float* h_data;
	int* h_dataSizes;
	double* h_dataTimes;

	float* d_data;
	int* d_dataSizes;
	double* d_dataTimes;
	double* d_params;
	double* d_initialConditions;

	hipMalloc((void**)& d_params, in_amountOfParams * sizeof(double));
	hipMalloc((void**)& d_initialConditions, in_amountOfParams * sizeof(double));
	hipMemcpy(d_params, in_params, in_amountOfParams * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(d_initialConditions, in_initialConditions, in_amountOfParams * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);

	size_t amountOfIteration = (size_t)std::ceilf((double)in_nPts / (double)nPtsLimiter);;

	std::ofstream outFileStream;
	outFileStream.open(in_outPath);

	for (size_t i = 0; i < amountOfIteration; ++i)
	{
		if (i == amountOfIteration - 1)
		{
			h_dataTimes = (double*)malloc((in_nPts - nPtsLimiter * i) * sizeof(double));

			slice(globalParamValues, nPtsLimiter * i, in_nPts, h_dataTimes);
			nPtsLimiter = in_nPts - (nPtsLimiter * i);
		}
		else
		{
			h_dataTimes = (double*)malloc(((nPtsLimiter * i + nPtsLimiter) - nPtsLimiter * i) * sizeof(double));
			slice(globalParamValues, nPtsLimiter * i, nPtsLimiter * i + nPtsLimiter, h_dataTimes);
		}


		h_data = (float*)malloc(nPtsLimiter * amountOfTPoints * sizeof(float));
		h_dataSizes = (int*)malloc(nPtsLimiter * sizeof(int));

		hipMalloc((void**)& d_data, nPtsLimiter * amountOfTPoints * sizeof(float));
		hipMalloc((void**)& d_dataSizes, nPtsLimiter * sizeof(int));
		hipMalloc((void**)& d_dataTimes, nPtsLimiter * sizeof(double));

		hipMemcpy(d_dataTimes, h_dataTimes, nPtsLimiter * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);

		int blockSize;
		int minGridSize;
		int gridSize;

		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, bifuractionKernel, 0, nPtsLimiter);
		gridSize = (nPtsLimiter + blockSize - 1) / blockSize;



		//Call CUDA func
		bifuractionKernel << <gridSize, blockSize >> > (nPtsLimiter,
			in_tMax,
			in_h,
			d_initialConditions,
			in_nValue,
			in_prePeakFinderSliceK,
			d_data,
			d_dataSizes,
			PEAKFINDER_MODE,
			in_thresholdValueOfMaxSignalValue,
			in_amountOfParams,
			in_discreteModelMode,
			in_prescaller,
			d_params,
			d_dataTimes,
			in_mode);


		hipMemcpy(h_data, d_data, amountOfTPoints * nPtsLimiter * sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);
		hipMemcpy(h_dataSizes, d_dataSizes, nPtsLimiter * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

		hipDeviceSynchronize();

		hipFree(d_data);
		hipFree(d_dataSizes);
		hipFree(d_dataTimes);

		for (size_t i = 0; i < nPtsLimiter; ++i)
			for (size_t j = 0; j < h_dataSizes[i]; ++j)
				if (outFileStream.is_open())
				{
					outFileStream << h_dataTimes[i] << ", " << h_data[i * amountOfTPoints + j] << '\n';
				}
				else
				{
					std::cout << "\nOutput file open error" << std::endl;
					exit(1);
				}

		std::free(h_data);
		std::free(h_dataSizes);
		std::free(h_dataTimes);

		if (in_debug)
			std::cout << "       " << std::setprecision(3) << (100.0f / (double)amountOfIteration) * (i + 1) << "%\n";

		progress.store((100.0f / (double)amountOfIteration) * (i + 1), std::memory_order_seq_cst);
	}

	if (in_debug)
	{
		if (amountOfIteration != 1)
			std::cout << "       " << "100%\n";
		std::cout << '\n';
	}

	hipFree(d_params);
	hipFree(d_initialConditions);
	std::free(globalParamValues);

	progress.store(100, std::memory_order_seq_cst);

	outFileStream.close();

	return;
}



__host__ void bifurcation2D(
	double					in_tMax,
	int					in_nPts,
	double				in_h,
	double* in_initialConditions,
	double				in_paramValues1,
	double				in_paramValues2,
	double				in_paramValues3,
	double				in_paramValues4,
	int					in_nValue,
	double				in_prePeakFinderSliceK,
	int					in_thresholdValueOfMaxSignalValue,
	int					in_amountOfParams,
	int					in_discreteModelMode,
	int					in_prescaller,
	double* in_params,
	int					in_mode1,
	int					in_mode2,
	int					in_kdeSampling,
	float				in_kdeSamplesInterval1,
	float				in_kdeSamplesInterval2,
	float				in_kdeSamplesSmooth,
	double				in_memoryLimit,
	std::string			in_outPath,
	bool				in_debug,
	std::atomic<int> & progress)
{
	std::ofstream outFileStream;
	outFileStream.open(in_outPath);
	outFileStream << in_paramValues1 << ", " << in_paramValues2 << "\n" << in_paramValues3 << ", " << in_paramValues4 << "\n";

	size_t amountOfTPoints = in_tMax / in_h / in_prescaller;

	double* paramValues1 = nullptr;
	double* paramValues2 = nullptr;

	paramValues1 = (double*)malloc(sizeof(double) * in_nPts * in_nPts);
	paramValues2 = (double*)malloc(sizeof(double) * in_nPts * in_nPts);

	getParamsAndSymmetry2D(paramValues1, paramValues2,
		in_paramValues1, in_paramValues2,
		in_paramValues3, in_paramValues4,
		in_nPts);

	size_t freeMemory;
	size_t totalMemory;

	hipMemGetInfo(&freeMemory, &totalMemory);
	//freeMemory = 7472152576;
	freeMemory *= in_memoryLimit * 0.95;

	double maxMemoryLimit = sizeof(double) * (amountOfTPoints + 2 + in_amountOfParams) + sizeof(int);

	size_t nPtsLimiter = freeMemory / maxMemoryLimit;

	if (nPtsLimiter <= 0)
	{
		if (in_debug)
			std::cout << "\nVery low memory size. Increase the MEMORY_LIMIT!" << "\n";
		exit(1);
	}

	int* h_kdeResult;
	float* h_data;
	double* h_paramValues1;
	double* h_paramValues2;

	double* d_params;
	int* d_kdeResult;
	float* d_data;
	double* d_paramValues1;
	double* d_paramValues2;
	double* d_initialConditions;


	hipMalloc((void**)& d_params, in_amountOfParams * sizeof(double));
	hipMalloc((void**)& d_initialConditions, in_amountOfParams * sizeof(double));

	hipMemcpy(d_params, in_params, in_amountOfParams * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(d_initialConditions, in_initialConditions, in_amountOfParams * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);

	size_t amountOfIteration = (size_t)std::ceilf((double)(in_nPts * in_nPts) / (double)nPtsLimiter);

	int stringCounter = 0;

	for (size_t i = 0; i < amountOfIteration; ++i)
	{
		if (i == amountOfIteration - 1)
		{
			h_kdeResult = (int*)malloc(((in_nPts * in_nPts) - nPtsLimiter * i) * sizeof(int));
			h_paramValues1 = (double*)malloc(((in_nPts * in_nPts) - nPtsLimiter * i) * sizeof(double));
			h_paramValues2 = (double*)malloc(((in_nPts * in_nPts) - nPtsLimiter * i) * sizeof(double));

			slice(paramValues1, nPtsLimiter * i, (in_nPts * in_nPts), h_paramValues1);
			slice(paramValues2, nPtsLimiter * i, (in_nPts * in_nPts), h_paramValues2);
			nPtsLimiter = (in_nPts * in_nPts) - (nPtsLimiter * i);
		}
		else
		{
			h_kdeResult = (int*)malloc(((nPtsLimiter * i + nPtsLimiter) - nPtsLimiter * i) * sizeof(int));
			h_paramValues1 = (double*)malloc(((nPtsLimiter * i + nPtsLimiter) - nPtsLimiter * i) * sizeof(double));
			h_paramValues2 = (double*)malloc(((nPtsLimiter * i + nPtsLimiter) - nPtsLimiter * i) * sizeof(double));

			slice(paramValues1, nPtsLimiter * i, nPtsLimiter * i + nPtsLimiter, h_paramValues1);
			slice(paramValues2, nPtsLimiter * i, nPtsLimiter * i + nPtsLimiter, h_paramValues2);
		}


		h_data = (float*)malloc(nPtsLimiter * amountOfTPoints * sizeof(float));

		hipMalloc((void**)& d_kdeResult, nPtsLimiter * sizeof(int));
		hipMalloc((void**)& d_data, nPtsLimiter * amountOfTPoints * sizeof(double));
		hipMalloc((void**)& d_paramValues1, nPtsLimiter * sizeof(double));
		hipMalloc((void**)& d_paramValues2, nPtsLimiter * sizeof(double));

		hipMemcpy(d_paramValues1, h_paramValues1, nPtsLimiter * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_paramValues2, h_paramValues2, nPtsLimiter * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);

		int blockSize;
		int minGridSize;
		int gridSize;

		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, bifuractionKernel, 0, nPtsLimiter);
		gridSize = (nPtsLimiter + blockSize - 1) / blockSize;

		//Call CUDA func
		bifuractionKernel << <gridSize, blockSize >> > (
			nPtsLimiter,
			in_tMax,
			in_h,
			d_initialConditions,
			in_nValue,
			in_prePeakFinderSliceK,
			d_data,
			d_kdeResult,
			KDE_MODE,
			in_thresholdValueOfMaxSignalValue,
			in_amountOfParams,
			in_discreteModelMode,
			in_prescaller,
			d_params,
			d_paramValues1,
			in_mode1,
			d_paramValues2,
			in_mode2,
			nullptr,
			0,
			in_kdeSampling,
			in_kdeSamplesInterval1,
			in_kdeSamplesInterval2,
			in_kdeSamplesSmooth);


		//hipMemcpy(h_data, d_data, amountOfTPoints * nPtsLimiter * sizeof(double), hipMemcpyKind::hipMemcpyDeviceToHost);
		hipMemcpy(h_kdeResult, d_kdeResult, nPtsLimiter * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

		hipDeviceSynchronize();

		hipFree(d_data);
		hipFree(d_kdeResult);
		hipFree(d_paramValues1);
		hipFree(d_paramValues2);

		for (size_t i = 0; i < nPtsLimiter; ++i)
			if (outFileStream.is_open())
			{
				if (stringCounter != 0)
					outFileStream << ", ";
				if (stringCounter == in_nPts)
				{
					outFileStream << "\n";
					stringCounter = 0;
				}
				outFileStream << h_kdeResult[i];
				++stringCounter;
			}
			else
			{
				std::cout << "\nOutput file open error" << std::endl;
				exit(1);
			}

		std::free(h_kdeResult);
		std::free(h_data);
		std::free(h_paramValues1);
		std::free(h_paramValues2);

		if (in_debug)
			std::cout << "       " << std::setprecision(3) << (100.0f / (double)amountOfIteration) * (i + 1) << "%\n";

		progress.store((100.0f / (double)amountOfIteration) * (i + 1), std::memory_order_seq_cst);
	}

	hipFree(d_params);
	hipFree(d_initialConditions);

	if (in_debug)
	{
		if (amountOfIteration != 1)
			std::cout << "       " << "100%\n";
		std::cout << '\n';
	}

	progress.store(100, std::memory_order_seq_cst);

	outFileStream.close();

	return;
}



__host__ void bifurcation3D(
	double					in_tMax,
	int					in_nPts,
	double				in_h,
	double* in_initialConditions,
	double				in_paramValues1,
	double				in_paramValues2,
	double				in_paramValues3,
	double				in_paramValues4,
	double				in_paramValues5,
	double				in_paramValues6,
	int					in_nValue,
	double				in_prePeakFinderSliceK,
	int					in_thresholdValueOfMaxSignalValue,
	int					in_amountOfParams,
	int					in_discreteModelMode,
	int					in_prescaller,
	double* in_params,
	int					in_mode1,
	int					in_mode2,
	int					in_mode3,
	int					in_kdeSampling,
	float				in_kdeSamplesInterval1,
	float				in_kdeSamplesInterval2,
	float				in_kdeSamplesSmooth,
	double				in_memoryLimit,
	std::string			in_outPath,
	bool				in_debug,
	std::atomic<int> & progress)
{
	std::ofstream outFileStream;
	outFileStream.open(in_outPath);
	outFileStream << in_paramValues1 << ", " << in_paramValues2 << "\n" << in_paramValues3 << ", " << in_paramValues4 << "\n" << in_paramValues5 << ", " << in_paramValues6 << "\n";

	size_t amountOfTPoints = in_tMax / in_h / in_prescaller;

	double* paramValues1 = nullptr;
	double* paramValues2 = nullptr;
	double* paramValues3 = nullptr;

	paramValues1 = (double*)malloc(sizeof(double) * in_nPts * in_nPts * in_nPts);
	paramValues2 = (double*)malloc(sizeof(double) * in_nPts * in_nPts * in_nPts);
	paramValues3 = (double*)malloc(sizeof(double) * in_nPts * in_nPts * in_nPts);

	getParamsAndSymmetry3D(paramValues1, paramValues2, paramValues3,
		in_paramValues1, in_paramValues2,
		in_paramValues3, in_paramValues4,
		in_paramValues5, in_paramValues6,
		in_nPts);

	size_t freeMemory;
	size_t totalMemory;

	hipMemGetInfo(&freeMemory, &totalMemory);
	//freeMemory = 7472152576;
	freeMemory *= in_memoryLimit * 0.95;

	double maxMemoryLimit = sizeof(double) * ((amountOfTPoints)+3 + in_amountOfParams) + sizeof(int);

	size_t nPtsLimiter = freeMemory / maxMemoryLimit;

	if (nPtsLimiter <= 0)
	{
		if (in_debug)
			std::cout << "\nVery low memory size. Increase the MEMORY_LIMIT!" << "\n";
		exit(1);
	}

	int* h_kdeResult;
	float* h_data;
	double* h_paramValues1;
	double* h_paramValues2;
	double* h_paramValues3;

	double* d_params;
	int* d_kdeResult;
	float* d_data;
	double* d_paramValues1;
	double* d_paramValues2;
	double* d_paramValues3;
	double* d_initialConditions;


	hipMalloc((void**)& d_params, in_amountOfParams * sizeof(double));
	hipMalloc((void**)& d_initialConditions, in_amountOfParams * sizeof(double));

	hipMemcpy(d_params, in_params, in_amountOfParams * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(d_initialConditions, in_initialConditions, in_amountOfParams * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);

	size_t amountOfIteration = (size_t)std::ceilf((double)(in_nPts * in_nPts * in_nPts) / (double)nPtsLimiter);

	int stringCounter = 0;

	for (size_t i = 0; i < amountOfIteration; ++i)
	{
		if (i == amountOfIteration - 1)
		{
			h_kdeResult = (int*)malloc(((in_nPts * in_nPts * in_nPts) - nPtsLimiter * i) * sizeof(int));
			h_paramValues1 = (double*)malloc(((in_nPts * in_nPts * in_nPts) - nPtsLimiter * i) * sizeof(double));
			h_paramValues2 = (double*)malloc(((in_nPts * in_nPts * in_nPts) - nPtsLimiter * i) * sizeof(double));
			h_paramValues3 = (double*)malloc(((in_nPts * in_nPts * in_nPts) - nPtsLimiter * i) * sizeof(double));

			slice(paramValues1, nPtsLimiter * i, (in_nPts * in_nPts * in_nPts), h_paramValues1);
			slice(paramValues2, nPtsLimiter * i, (in_nPts * in_nPts * in_nPts), h_paramValues2);
			slice(paramValues3, nPtsLimiter * i, (in_nPts * in_nPts * in_nPts), h_paramValues3);
			nPtsLimiter = (in_nPts * in_nPts * in_nPts) - (nPtsLimiter * i);
		}
		else
		{
			h_kdeResult = (int*)malloc(((nPtsLimiter * i + nPtsLimiter) - nPtsLimiter * i) * sizeof(int));
			h_paramValues1 = (double*)malloc(((nPtsLimiter * i + nPtsLimiter) - nPtsLimiter * i) * sizeof(double));
			h_paramValues2 = (double*)malloc(((nPtsLimiter * i + nPtsLimiter) - nPtsLimiter * i) * sizeof(double));
			h_paramValues3 = (double*)malloc(((nPtsLimiter * i + nPtsLimiter) - nPtsLimiter * i) * sizeof(double));

			slice(paramValues1, nPtsLimiter * i, nPtsLimiter * i + nPtsLimiter, h_paramValues1);
			slice(paramValues2, nPtsLimiter * i, nPtsLimiter * i + nPtsLimiter, h_paramValues2);
			slice(paramValues3, nPtsLimiter * i, nPtsLimiter * i + nPtsLimiter, h_paramValues3);
		}


		h_data = (float*)malloc(nPtsLimiter * amountOfTPoints * sizeof(float));

		hipMalloc((void**)& d_kdeResult, nPtsLimiter * sizeof(int));
		hipMalloc((void**)& d_data, nPtsLimiter * amountOfTPoints * sizeof(double));
		hipMalloc((void**)& d_paramValues1, nPtsLimiter * sizeof(double));
		hipMalloc((void**)& d_paramValues2, nPtsLimiter * sizeof(double));
		hipMalloc((void**)& d_paramValues3, nPtsLimiter * sizeof(double));

		hipMemcpy(d_paramValues1, h_paramValues1, nPtsLimiter * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_paramValues2, h_paramValues2, nPtsLimiter * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_paramValues3, h_paramValues3, nPtsLimiter * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);

		int blockSize;
		int minGridSize;
		int gridSize;

		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, bifuractionKernel, 0, nPtsLimiter);
		gridSize = (nPtsLimiter + blockSize - 1) / blockSize;

		//Call CUDA func
		bifuractionKernel << <gridSize, blockSize >> > (
			nPtsLimiter,
			in_tMax,
			in_h,
			d_initialConditions,
			in_nValue,
			in_prePeakFinderSliceK,
			d_data,
			d_kdeResult,
			KDE_MODE,
			in_thresholdValueOfMaxSignalValue,
			in_amountOfParams,
			in_discreteModelMode,
			in_prescaller,
			d_params,
			d_paramValues1,
			in_mode1,
			d_paramValues2,
			in_mode2,
			d_paramValues3,
			in_mode3,
			in_kdeSampling,
			in_kdeSamplesInterval1,
			in_kdeSamplesInterval2,
			in_kdeSamplesSmooth);


		//hipMemcpy(h_data, d_data, amountOfTPoints * nPtsLimiter * sizeof(double), hipMemcpyKind::hipMemcpyDeviceToHost);
		hipMemcpy(h_kdeResult, d_kdeResult, nPtsLimiter * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

		hipDeviceSynchronize();

		hipFree(d_data);
		hipFree(d_kdeResult);
		hipFree(d_paramValues1);
		hipFree(d_paramValues2);
		hipFree(d_paramValues3);

		for (size_t i = 0; i < nPtsLimiter; ++i)
			if (outFileStream.is_open())
			{
				if (stringCounter != 0)
					outFileStream << ", ";
				if (stringCounter == in_nPts)
				{
					outFileStream << "\n";
					stringCounter = 0;
				}
				outFileStream << h_kdeResult[i];
				++stringCounter;
			}
			else
			{
				std::cout << "\nOutput file open error" << std::endl;
				exit(1);
			}

		std::free(h_kdeResult);
		std::free(h_data);
		std::free(h_paramValues1);
		std::free(h_paramValues2);
		std::free(h_paramValues3);

		if (in_debug)
			std::cout << "       " << std::setprecision(3) << (100.0f / (double)amountOfIteration) * (i + 1) << "%\n";

		progress.store((100.0f / (double)amountOfIteration) * (i + 1), std::memory_order_seq_cst);
	}

	hipFree(d_params);
	hipFree(d_initialConditions);

	if (in_debug)
	{
		if (amountOfIteration != 1)
			std::cout << "       " << "100%\n";
		std::cout << '\n';
	}

	progress.store(100, std::memory_order_seq_cst);

	outFileStream.close();

	return;
}



__global__ void bifuractionKernel(
	int in_nPts,
	double in_TMax,
	double in_h,
	double* in_initialConditions,
	int in_nValue,
	double in_prePeakFinderSliceK,
	float* in_data,
	int* in_dataSizes,
	ResultMode resultMode,
	int thresholdValueOfMaxSignalValue,
	int	in_amountOfParams,
	int in_discreteModelMode,
	int	in_prescaller,
	double* in_params,
	double* in_paramValues1,
	int	in_mode1,
	double* in_paramValues2,
	int in_mode2,
	double* in_paramValues3,
	int in_mode3,
	int in_kdeSampling,
	float in_kdeSamplesInterval1,
	float in_kdeSamplesInterval2,
	float in_kdeSmoothH
)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= in_nPts)
		return;

	size_t amountOfTPoints = in_TMax / in_h / in_prescaller;
	size_t amountOfSkipPoints = in_prePeakFinderSliceK / in_h;
	size_t index = amountOfTPoints * idx;
	// Change to dynamic / KISH
	//double x[3]{ in_initialConditions[0], in_initialConditions[1], in_initialConditions[2] };
	double x[4]{ in_initialConditions[0], in_initialConditions[1], in_initialConditions[2], in_initialConditions[3] };


	double* localParam = new double[in_amountOfParams];
	for (int i = 0; i < in_amountOfParams; ++i)
		localParam[i] = in_params[i];

	localParam[in_mode1] = in_paramValues1[idx];

	if (in_paramValues2 != nullptr)
		localParam[in_mode2] = in_paramValues2[idx];

	if (in_paramValues3 != nullptr)
		localParam[in_mode3] = in_paramValues3[idx];

	//double localH1 = in_h * localParam[0];
	//double localH2 = in_h * (1 - localParam[0]);

	//Skip PREPEAKFINDER points
	for (size_t i = 0; i < amountOfSkipPoints; ++i) {

		calculateDiscreteModel(in_discreteModelMode, x, localParam, in_h);

		if (resultMode == KDE_MODE && abs(x[in_nValue]) > thresholdValueOfMaxSignalValue)
		{
			in_dataSizes[idx] = 0;
			delete[] localParam;
			return;
		}

		if (resultMode == PEAKFINDER_MODE && abs(x[in_nValue]) > thresholdValueOfMaxSignalValue)
		{
			in_dataSizes[idx] = 0;
			delete[] localParam;
			return;
		}

	}

	//Calculating
	for (size_t i = 0; i < amountOfTPoints; ++i)
	{
		in_data[index + i] = (float)(x[in_nValue]);
		for (size_t j = 0; j < in_prescaller - 1; ++j)
			calculateDiscreteModel(in_discreteModelMode, x, localParam, in_h);
		calculateDiscreteModel(in_discreteModelMode, x, localParam, in_h);

		if (resultMode == KDE_MODE && abs(x[in_nValue]) > thresholdValueOfMaxSignalValue)
		{
			in_dataSizes[idx] = 0;
			delete[] localParam;
			return;
		}

		if (resultMode == PEAKFINDER_MODE && abs(x[in_nValue]) > thresholdValueOfMaxSignalValue)
		{
			in_dataSizes[idx] = 0;
			delete[] localParam;
			return;
		}
	}

	delete[] localParam;

	// Here is the choice of method: KDE or peakFinder
	// TODO add switch on method of result
	// WARNING!!! THIS METHOD MUST TO FILL dataSizes[] (IF NEEDED)!!!

	int outSize = 0;

	switch (resultMode)
	{
	case PEAKFINDER_MODE:
		//		peakFinder(idx, 0, amountOfTPoints, in_data, in_dataSizes, in_data);
		peakFinderForDBSCAN(idx, (float)in_h, (float)0, amountOfTPoints, in_data, in_data, in_dataSizes);
		break;
	case KDE_MODE:
		//outSize = peakFinder(idx, 0, amountOfTPoints, in_data, in_dataSizes, in_data);
		//in_dataSizes[idx] = outSize;
		//kdeMethod(idx, in_data, in_dataSizes, in_kdeSampling, outSize, in_kdeSamplesInterval1, in_kdeSamplesInterval2, amountOfTPoints, in_kdeSmoothH, amountOfTPoints*0.1);


		outSize = peakFinderForDBSCAN(idx, in_h, 0, amountOfTPoints, in_data, in_data, in_dataSizes);

		//float maxx = -9999;//0.002
		//float maxy = -9999; //0.1
		//float minx = 9999;//0.002
		//float miny = 9999; //0.1

		//for (int i = 0; i < outSize; i++) {
		//	if (in_data[index + i * 2] > maxx) {
		//		maxx = in_data[index + i * 2];
		//	}
		//	if (in_data[index + i * 2 + 1] > maxy) {
		//		maxy = in_data[index + i * 2];
		//	}
		//	if (in_data[index + i * 2] < minx) {
		//		minx = in_data[index + i * 2];
		//	}
		//	if (in_data[index + i * 2 + 1] < miny) {
		//		miny = in_data[index + i * 2];
		//	}
		//}


		//maxx = 1 / (maxx - minx);
		//maxy = 1 / (maxy - miny);
		////maxx = 0;
		////maxy = 0;
		////float maxx = 1;
		////float maxy = 0;
		//for (int i = 0; i < outSize; i++) {
		//	in_data[index + i * 2] = (in_data[index + i * 2]) * maxx;
		//	in_data[index + i * 2 + 1] = in_data[index + i * 2 + 1] * maxy;
		//}

		//float maxx = 2;//0.002
		//float maxy = 40; //0.1
		//float minx = -2;//0.002
		//float miny = 0; //0.1
		//maxx = 1 / (maxx - minx);
		//maxy = 1 / (maxy - miny);
		//for (int i = 0; i < outSize; i++) {
		//	in_data[index + i * 2] = (in_data[index + i * 2]) * maxx;
		//	in_data[index + i * 2 + 1] = in_data[index + i * 2 + 1] * maxy;
		//}

		//float maxx = 1;//0.002
		//float maxy = 0; //0.1
		//for (int i = 0; i < outSize; i++) {
		//	in_data[index + i * 2] = (in_data[index + i * 2]) * maxx;
		//	in_data[index + i * 2 + 1] = in_data[index + i * 2 + 1] * maxy;
		//}


		float maxx = 11;//0.002
		float maxy = 8; //0.1
		float minx = -1.5;//0.002
		float miny = 0; //0.1
		float deltx = 1 / (maxx - minx);
		float delty = 1 / (maxy - miny);
		for (int i = 0; i < outSize; i++) {
			in_data[index + i * 2] = (in_data[index + i * 2] - minx) * deltx;
			in_data[index + i * 2 + 1] = (in_data[index + i * 2 + 1] - miny) * delty;
		}

		dbscan(in_data, amountOfTPoints, outSize, idx, 0.01f, in_dataSizes, 0.2 * amountOfTPoints);
//		in_dataSizes[idx] = (int)(miny*1000);
		break;
	}
}


__device__ void calculateDiscreteModel(int mode, double* X, double* a, double h)
{


	switch (mode)
	{
	case ROSSLER: // 0.2 0.2 5.7
		//x[0] = x[0] + localH1 * (-x[1] - x[2]);
		//x[1] = (x[1] + localH1 * (x[0])) / (1 - values[1] * localH1);
		//x[2] = (x[2] + localH1 * values[2]) / (1 - localH1 * (x[0] - values[3]));
		//x[2] = x[2] + localH2 * (values[2] + x[2] * (x[0] - values[3]));
		//x[1] = x[1] + localH2 * (x[0] + values[1] * x[1]);
		//x[0] = x[0] + localH2 * (-x[1] - x[2]);
		break;
	case CHEN: // 40 3 28
		//x[0] = (x[0] + localH1 * values[1] * x[1]) / (1 + localH1 * values[1]);
		//x[1] = (x[1] + localH1 * x[0] * (values[3] - values[1] - x[2])) / (1 - localH1 * values[3]);
		//x[2] = (x[2] + localH1 * x[0] * x[1]) / (1 + localH1 * values[2]);
		//x[2] = x[2] + localH2 * (x[0] * x[1] - values[2] * x[2]);
		//x[1] = x[1] + localH2 * (x[0] * (values[3] - values[1] - x[2]) + values[3] * x[1]);
		//x[0] = x[0] + localH2 * (values[1] * (x[1] - x[0]));
		break;
	case LORENZ: // 10 28 2.6667
		//x[0] = (x[0] + localH1 * values[1] * x[1]) / (1 + localH1 * values[1]);
		//x[1] = (x[1] + localH1 * x[0] * (values[2] - x[2])) / (1 + localH1);
		//x[2] = (x[2] + localH1 * x[0] * x[1]) / (1 + localH1 * values[3]);
		//x[2] = x[2] + localH2 * (x[0] * x[1] - values[3] * x[2]);
		//x[1] = x[1] + localH2 * (x[0] * (values[2] - x[2]) - x[1]);
		//x[0] = x[0] + localH2 * (values[1] * (x[1] - x[0])); 
		break;
	case LORENZ_RYBIN: // -2,5 10 20 3 -0,695
		//x[0] = x[0] + localH1 * (values[2] * x[1] - values[1] * x[0] + values[5] * x[1] * x[2]);
		//x[1] = x[1] + localH1 * (values[3] * x[0] - x[0] * x[2] - x[1]);
		//x[2] = x[2] + localH1 * (x[0] * x[1] - values[4] * x[2]);
		//x[2] = (x[2] + localH2 * (x[0] * x[1])) / (1 + values[4] * localH2);
		//x[1] = (x[1] + localH2 * (values[3] * x[0] - x[0] * x[2])) / (1 + localH2);
		//x[0] = (x[0] + localH2 * (values[2] * x[1] + values[5] * x[1] * x[2])) / (1 + localH2 * values[1]);
		break;
	case CONSERVA: // -2,5 10 20 3 -0,695
		//double h_local = h * 1.35120719196;
		//double h1 = h_local * a[0];
		//double h2 = h_local * (1 - a[0]);4

		//double h_local = h * 0.5;
		//double h1 = h_local * 1.35120719196;
		//double h2 = h1;
		//X[0] = X[0] + h1 * (X[1] + X[0] * X[2]);
		//X[1] = X[1] + h1 * (-a[2] * X[0] + X[1] * X[2] + X[3]);
		//X[2] = X[2] + h1 * (1 - X[0] * X[0] - X[1] * X[1]);
		//X[3] = X[3] + h1 * (-a[1] * X[1]);
		//X[3] = X[3] + h2 * (-a[1] * X[1]);
		//X[2] = X[2] + h2 * (1 - X[0] * X[0] - X[1] * X[1]);
		//X[1] = (X[1] + h2 * (-a[2] * X[0] + X[3])) / (1 - h2 * X[2]);
		//X[0] = (X[0] + h2 * (X[1])) / (1 - h2 * X[2]);
		////h_local = h * (-1.702414383919);
		//h1 = h_local * (-1.702414383919);
		//h2 = h1;
		//X[0] = X[0] + h1 * (X[1] + X[0] * X[2]);
		//X[1] = X[1] + h1 * (-a[2] * X[0] + X[1] * X[2] + X[3]);
		//X[2] = X[2] + h1 * (1 - X[0] * X[0] - X[1] * X[1]);
		//X[3] = X[3] + h1 * (-a[1] * X[1]);
		//X[3] = X[3] + h2 * (-a[1] * X[1]);
		//X[2] = X[2] + h2 * (1 - X[0] * X[0] - X[1] * X[1]);
		//X[1] = (X[1] + h2 * (-a[2] * X[0] + X[3])) / (1 - h2 * X[2]);
		//X[0] = (X[0] + h2 * (X[1])) / (1 - h2 * X[2]);
		//h1 = h_local * 1.35120719196;
		//h2 = h1;
		//X[0] = X[0] + h1 * (X[1] + X[0] * X[2]);
		//X[1] = X[1] + h1 * (-a[2] * X[0] + X[1] * X[2] + X[3]);
		//X[2] = X[2] + h1 * (1 - X[0] * X[0] - X[1] * X[1]);
		//X[3] = X[3] + h1 * (-a[1] * X[1]);
		//X[3] = X[3] + h2 * (-a[1] * X[1]);
		//X[2] = X[2] + h2 * (1 - X[0] * X[0] - X[1] * X[1]);
		//X[1] = (X[1] + h2 * (-a[2] * X[0] + X[3])) / (1 - h2 * X[2]);
		//X[0] = (X[0] + h2 * (X[1])) / (1 - h2 * X[2]);

		//double X1[4];
		//double k[4][4];
		//int N = 4;
		//int i, j;
		//for (i = 0; i < N; i++) {
		//	X1[i] = X[i];
		//}
		//for (j = 0; j < 4; j++) {
		//	k[0][j] = (X1[1] + X1[0] * X1[2]);
		//	k[1][j] = (-a[2] * X1[0] + X1[1] * X1[2] + X1[3]);
		//	k[2][j] = (1 - X1[0] * X1[0] - X1[1] * X1[1]);
		//	k[3][j] = (-a[1] * X1[1]);
		//	if (j == 3) {
		//		for (i = 0; i < N; i++) {
		//			X[i] = X[i] + h * (k[i][0] + 2 * k[i][1] + 2 * k[i][2] + k[i][3]) / 6;
		//		}
		//	}
		//	else if (j == 2) {
		//		for (i = 0; i < N; i++) {
		//			X1[i] = X[i] + h * k[i][j];
		//		}
		//	}
		//	else {
		//		for (i = 0; i < N; i++) {
		//			X1[i] = X[i] + 0.5 * h * k[i][j];
		//		}
		//	}
		//}

		//int M = 4;
		//int N = 4;
		//double X1[4];
		//double k[4][4];
		//double A[4][4];
		//double B[4];
		//int N = 4;
		//int i, j, l;
		//for (i = 0; i < N; i++) {
		//	for (j = 0; i <= i - 1 ; i++) {
		//		for (l = 0; i < M; i++) {
		//			k[l][i] = k[l][i] + k[l][j] * A[i][j];
		//		}
		//	}
		//	for (l = 0; i < M; i++) {
		//		k[l][i] = k[l][i]*h + X[l];
		//	}
		//	k[0][i] = (k[1][i] + k[0][i] * k[2][i]);
		//	k[1][i] = (-a[2] * k[0][i] + k[1][i] * k[2][i] + k[3][i]);
		//	k[2][j] = (1 - k[0][i] * k[0][i] - k[1][i] * k[1][i]);
		//	k[3][j] = (-a[1] * k[1][i]);
		//}
		//for (i = 0; i < N; i++) {
		//	for (l = 0; i < M; i++) {
		//		X[l] = X[l] + h * B[i] * k[l][i];
		//	}
		//}


		break;
	case DISSIPATA:
		////jOSEPHSON jUNCTION double params[6]{ 2.7,0.75,1.2,6.9,0.367,0.0478 };
		//double X1[3];
		//h = h * 0.5;
		//X[0] = X[0] + h * (X[1]);
		//X[1] = X[1] + h * ((1 / a[1]) * (a[2] - ((X[1] > a[3]) ? a[4] : a[5]) * X[1] - sinf( (float)X[0]) - X[2]));
		//X[2] = X[2] + h * ((1 / a[0]) * (X[1] - X[2]));
		//X1[0] = X[0];
		//X1[1] = X[1];
		//X1[2] = X[2];
		//X[2] = (X1[2] + h * (1 / a[0]) * X[1]) / (1 + h * (1 / a[0]));
		//X[1] = X1[1] + h * ((1 / a[1]) * (a[2] - ((X[1] > a[3]) ? a[4] : a[5]) * X[1] - sinf((float)X[0]) - X[2]));
		//X[1] = X1[1] + h * ((1 / a[1]) * (a[2] - ((X[1] > a[3]) ? a[4] : a[5]) * X[1] - sinf((float)X[0]) - X[2]));
		//X[0] = X1[0] + h * (X[1]);

		//Rossler double params[6]{ 0.5,0.2,0.2,5.7 };
		//double X1[3];
		//h = 0.5 * h;
		//X1[0] = X[0] + h * (-X[1] - X[2]);
		//X1[1] = X[1] + h * (X[0] + a[1] * X[1]);
		//X1[2] = X[2] + h * (a[2] + X[2] * (X[0] - a[3]));
		//h = h * 2;
		//X[0] = X[0] + h * (-X1[1] - X1[2]);
		//X[1] = X[1] + h * (X1[0] + a[1] * X1[1]);
		//X[2] = X[2] + h * (a[2] + X1[2] * (X1[0] - a[3]));


		h = 0.5 * h;
		X[0] = X[0] + h * (-X[1] - X[2]);
		X[1] = (X[1] + h * (X[0])) / (1 - a[1] * h);
		X[2] = (X[2] + h * a[2]) / (1 - h * (X[0] - a[3]));
		X[2] = X[2] + h * (a[2] + X[2] * (X[0] - a[3]));
		X[1] = X[1] + h * (X[0] + a[1] * X[1]);
		X[0] = X[0] + h * (-X[1] - X[2]);


		/*
		double w[3][4];
		double h2 = h * 0.5;

		w[0][0] = 1 + a[1] * h2;
		w[0][1] = -h2 * a[1];
		w[0][2] = 0;
		w[1][0] = -h2 * (a[2] - X[2]);
		w[1][1] = 1 + 1 * h2;
		w[1][2] = h2 * X[0];
		w[2][0] = -h2 * X[1];
		w[2][1] = -h2 * X[0];
		w[2][2] = 1 + a[3] * h2;

		w[0][3] = h * a[1] * (X[1] - X[0]);
		w[1][3] = h * (a[2] * X[0] - X[1] - X[0] * X[2]);
		w[2][3] = h * (X[0] * X[1] - a[3] * X[2]);

		int HEIGHT = 3;
		int WIDTH = 4;
		int k; int i; int j; float t; float d;

		for (k = 0; k <= HEIGHT - 2; k++) {

			int l = k;

			for (i = k + 1; i <= HEIGHT - 1; i++) {
				if (abs(w[i][k]) > abs(w[l][k])) {
					l = i;
				}
			}
			if (l != k) {
				for (j = 0; j <= WIDTH - 1; j++) {
					if ((j == 0) || (j >= k)) {
						t = w[k][j];
						w[k][j] = w[l][j];
						w[l][j] = t;
					}
				}
			}

			d = 1.0 / w[k][k];
			for (i = (k + 1); i <= (HEIGHT - 1); i++) {
				if (w[i][k] == 0) {
					continue;
				}
				t = w[i][k] * d;
				for (j = k; j <= (WIDTH - 1); j++) {
					if (w[k][j] != 0) {
						w[i][j] = w[i][j] - t * w[k][j];
					}
				}
			}
		}

		for (i = (HEIGHT); i >= 2; i--) {
			for (j = 1; j <= i - 1; j++) {
				t = w[i - j - 1][i - 1] / w[i - 1][i - 1];
				w[i - j - 1][WIDTH - 1] = w[i - j - 1][WIDTH - 1] - t * w[i - 1][WIDTH - 1];
			}
			w[i - 1][WIDTH - 1] = w[i - 1][WIDTH - 1] / w[i - 1][i - 1];
		}
		w[0][WIDTH - 1] = w[0][WIDTH - 1] / w[0][0];

		X[0] = X[0] + w[0][WIDTH - 1];
		X[1] = X[1] + w[1][WIDTH - 1];
		X[2] = X[2] + w[2][WIDTH - 1];
		*/

		//double X1[3];
		//h = 0.5 * h;
		//X1[0] = X[0] + h * a[1] * (X[1] - X[0]);
		//X1[1] = X[1] + h * (X[0] * (a[2] - X[2]) - X[1]);
		//X1[2] = X[2] + h * (X[0] * X[1] - a[3] * X[2]);
		//h = h * 2;
		//X[0] = X[0] + h * a[1] * (X1[1] - X1[0]);
		//X[1] = X[1] + h * (X1[0] * (a[2] - X1[2]) - X1[1]);
		//X[2] = X[2] + h * (X1[0] * X1[1] - a[3] * X1[2]);

		////Lorenz
		//h = 0.5 * h;
		//X[0] = (X[0] + h * a[1] * X[1]) / (1 + h * a[1]);
		//X[1] = (X[1] + h * X[0] * (a[2] - X[2])) / (1 + h);
		//X[2] = (X[2] + h * X[0] * X[1]) / (1 + h * a[3]);
		//X[2] = X[2] + h * (X[0] * X[1] - a[3] * X[2]);
		//X[1] = X[1] + h * (X[0] * (a[2] - X[2]) - X[1]);
		//X[0] = X[0] + h * (a[1] * (X[1] - X[0]));


		//double h_local = h * 1.35120719196;
		//double h_local = h;
		//double h1 = h_local * a[0];
		//double h2 = h_local * (1 - a[0]);
		////Lorenz
		//X[0] = (X[0] + h1 * a[1] * X[1]) / (1 + h1 * a[1]);
		//X[1] = (X[1] + h1 * X[0] * (a[2] - X[2])) / (1 + h1);
		//X[2] = (X[2] + h1 * X[0] * X[1]) / (1 + h1 * a[3]);
		//X[2] = X[2] + h2 * (X[0] * X[1] - a[3] * X[2]);
		//X[1] = X[1] + h2 * (X[0] * (a[2] - X[2]) - X[1]);
		//X[0] = X[0] + h2 * (a[1] * (X[1] - X[0]));
		//h_local = h * (-1.702414383919);
		//h1 = h_local * a[0];
		//h2 = h_local * (1 - a[0]);
		//X[0] = (X[0] + h1 * a[1] * X[1]) / (1 + h1 * a[1]);
		//X[1] = (X[1] + h1 * X[0] * (a[2] - X[2])) / (1 + h1);
		//X[2] = (X[2] + h1 * X[0] * X[1]) / (1 + h1 * a[3]);
		//X[2] = X[2] + h2 * (X[0] * X[1] - a[3] * X[2]);
		//X[1] = X[1] + h2 * (X[0] * (a[2] - X[2]) - X[1]);
		//X[0] = X[0] + h2 * (a[1] * (X[1] - X[0]));
		//h_local = h * 1.35120719196;
		//h1 = h_local * a[0];
		//h2 = h_local * (1 - a[0]);
		//X[0] = (X[0] + h1 * a[1] * X[1]) / (1 + h1 * a[1]);
		//X[1] = (X[1] + h1 * X[0] * (a[2] - X[2])) / (1 + h1);
		//X[2] = (X[2] + h1 * X[0] * X[1]) / (1 + h1 * a[3]);
		//X[2] = X[2] + h2 * (X[0] * X[1] - a[3] * X[2]);
		//X[1] = X[1] + h2 * (X[0] * (a[2] - X[2]) - X[1]);
		//X[0] = X[0] + h2 * (a[1] * (X[1] - X[0]));


		/*double X1[3];
		double k[3][4];
		int N = 4;
		int i, j;
		for (i = 0; i < N; i++) {
			X1[i] = X[i];
		}
		for (j = 0; j < 4; j++) {
		k[0][j] = (a[1] * (X1[1] - X1[0]));
		k[1][j] = (X1[0] * (a[2] - X1[2]) - X1[1]);
		k[2][j] = (X1[0] * X1[1] - a[3] * X1[2]);
			if (j == 3) {
				for (i = 0; i < N; i++) {
					X1[i] = X[i] + h * (k[i][0] + 2 * k[i][1] + 2 * k[i][2] + k[i][3]) / 6;
				}
			}
			else if (j == 2) {
				for (i = 0; i < N; i++) {
					X1[i] = X[i] + h * k[i][j];
				}
			}
			else {
				for (i = 0; i < N; i++) {
					X1[i] = X[i] + 0.5 * h * k[i][j];
				}
			}
		}
		for (i = 0; i < N; i++) {
			X[i] = X1[i];
		}*/

		break;
	case TIMUR:
		//double X1[3];
		//double k[3][4];
		//int N = 3;
		//int i, j;
		//for (i = 0; i < N; i++) {
		//	X1[i] = X[i];
		//}
		//for (j = 0; j < 4; j++) {
		//	k[0][j] = (X1[2] - X1[0] * (a[1] + a[2])) * a[3] * a[4];
		//	k[1][j] = (a[5] * X1[0] * X1[2] / 10 * a[6] * a[7] - X1[1] * (a[8] + a[2])) * a[3] * a[4];
		//	k[2][j] = (-a[5] * X1[0] * a[9] + a[5] * X1[1] * a[10]) * a[11];

		//	if (j == 3) {
		//		for (i = 0; i < N; i++) {
		//			X1[i] = X[i] + h * (k[i][0] + 2 * k[i][1] + 2 * k[i][2] + k[i][3]) / 6;
		//		}
		//	}
		//	else if (j == 2) {
		//		for (i = 0; i < N; i++) {
		//			X1[i] = X[i] + h * k[i][j];
		//		}
		//	}
		//	else {
		//		for (i = 0; i < N; i++) {
		//			X1[i] = X[i] + 0.5 * h * k[i][j];
		//		}
		//	}
		//}
		//for (i = 0; i < N; i++) {
		//	X[i] = X1[i];
		//}
		//break;
	case CompCD:

		double h_local = h * 0.5;
		double h1 = h_local * 1.35120719196;
		double h2 = h1;
		X[0] = X[0] + h1 * (X[1] + X[0] * X[2]);
		X[1] = X[1] + h1 * (-a[2] * X[0] + X[1] * X[2] + X[3]);
		X[2] = X[2] + h1 * (1 - X[0] * X[0] - X[1] * X[1]);
		X[3] = X[3] + h1 * (-a[1] * X[1]);
		X[3] = X[3] + h2 * (-a[1] * X[1]);
		X[2] = X[2] + h2 * (1 - X[0] * X[0] - X[1] * X[1]);
		X[1] = (X[1] + h2 * (-a[2] * X[0] + X[3])) / (1 - h2 * X[2]);
		X[0] = (X[0] + h2 * (X[1])) / (1 - h2 * X[2]);
		//h_local = h * (-1.702414383919);
		h1 = h_local * (-1.702414383919);
		h2 = h1;
		X[0] = X[0] + h1 * (X[1] + X[0] * X[2]);
		X[1] = X[1] + h1 * (-a[2] * X[0] + X[1] * X[2] + X[3]);
		X[2] = X[2] + h1 * (1 - X[0] * X[0] - X[1] * X[1]);
		X[3] = X[3] + h1 * (-a[1] * X[1]);
		X[3] = X[3] + h2 * (-a[1] * X[1]);
		X[2] = X[2] + h2 * (1 - X[0] * X[0] - X[1] * X[1]);
		X[1] = (X[1] + h2 * (-a[2] * X[0] + X[3])) / (1 - h2 * X[2]);
		X[0] = (X[0] + h2 * (X[1])) / (1 - h2 * X[2]);
		h1 = h_local * 1.35120719196;
		h2 = h1;
		X[0] = X[0] + h1 * (X[1] + X[0] * X[2]);
		X[1] = X[1] + h1 * (-a[2] * X[0] + X[1] * X[2] + X[3]);
		X[2] = X[2] + h1 * (1 - X[0] * X[0] - X[1] * X[1]);
		X[3] = X[3] + h1 * (-a[1] * X[1]);
		X[3] = X[3] + h2 * (-a[1] * X[1]);
		X[2] = X[2] + h2 * (1 - X[0] * X[0] - X[1] * X[1]);
		X[1] = (X[1] + h2 * (-a[2] * X[0] + X[3])) / (1 - h2 * X[2]);
		X[0] = (X[0] + h2 * (X[1])) / (1 - h2 * X[2]);
		break;
	case RK4:
		double X1[4];
		double k[4][4];
		int N = 4;
		int i, j;
		for (i = 0; i < N; i++) {
			X1[i] = X[i];
		}
		for (j = 0; j < 4; j++) {
			k[0][j] = (X1[1] + X1[0] * X1[2]);
			k[1][j] = (-a[2] * X1[0] + X1[1] * X1[2] + X1[3]);
			k[2][j] = (1 - X1[0] * X1[0] - X1[1] * X1[1]);
			k[3][j] = (-a[1] * X1[1]);
			if (j == 3) {
				for (i = 0; i < N; i++) {
					X[i] = X[i] + h * (k[i][0] + 2 * k[i][1] + 2 * k[i][2] + k[i][3]) / 6;
				}
			}
			else if (j == 2) {
				for (i = 0; i < N; i++) {
					X1[i] = X[i] + h * k[i][j];
				}
			}
			else {
				for (i = 0; i < N; i++) {
					X1[i] = X[i] + 0.5 * h * k[i][j];
				}
			}
		}
		break;

	}
}

__device__ int peakFinder(int idx, float prePeakFinder, size_t amountOfTPoints, float* in_data, int* out_dataSizes, float* out_data)
{
	int _outSize = 0;
	for (size_t i = 1 + prePeakFinder * amountOfTPoints; i < amountOfTPoints - 1; ++i)
	{
		if (in_data[idx * amountOfTPoints + i] > in_data[idx * amountOfTPoints + i - 1] && in_data[idx * amountOfTPoints + i] > in_data[idx * amountOfTPoints + i + 1])
		{
			out_data[idx * amountOfTPoints + _outSize] = in_data[idx * amountOfTPoints + i];
			++_outSize;
		}
		else if (in_data[idx * amountOfTPoints + i] > in_data[idx * amountOfTPoints + i - 1] && in_data[idx * amountOfTPoints + i] == in_data[idx * amountOfTPoints + i + 1])
		{
			for (size_t k = i; k < amountOfTPoints - 1; ++k)
			{
				if (in_data[idx * amountOfTPoints + k] < in_data[idx * amountOfTPoints + k + 1])
				{
					break;
					i = k;
				}
				if (in_data[idx * amountOfTPoints + k] == in_data[idx * amountOfTPoints + k + 1])
					continue;
				if (in_data[idx * amountOfTPoints + k] > in_data[idx * amountOfTPoints + k + 1])
				{
					out_data[idx * amountOfTPoints + _outSize] = in_data[idx * amountOfTPoints + k];
					++_outSize;
					i = k + 1;
					break;
				}
			}
		}
	}

	out_dataSizes[idx] = _outSize;
	return _outSize;
}

__device__ int peakFinderForDBSCAN(int idx, float in_h, float prePeakFinder, size_t amountOfTPoints, float* in_data, float* out_data, int* out_dataSizes)
{
	size_t index = idx * amountOfTPoints;
	int _outSize = 0;

	for (int i = 3 + prePeakFinder * amountOfTPoints; i < amountOfTPoints - 1; ++i)
	{
		if (in_data[index + i] > in_data[index + i - 1] && in_data[index + i] > in_data[index + i + 1])
		{
			out_data[index + _outSize * 2] = in_data[index + i];
			out_data[index + _outSize * 2 + 1] = i;
			++_outSize;
		}
		else if (in_data[index + i] > in_data[index + i - 1] && in_data[index + i] == in_data[index + i + 1])
		{
			for (size_t k = i; k < amountOfTPoints - 1; ++k)
			{
				if (in_data[index + k] < in_data[index + k + 1])
				{
					break;
					i = k;
				}
				if (in_data[index + k] == in_data[index + k + 1])
					continue;
				if (in_data[index + k] > in_data[index + k + 1])
				{
					out_data[index + _outSize * 2] = in_data[index + k];
					out_data[index + _outSize * 2 + 1] = k;
					_outSize++;
					i = k + 1;
					break;
				}
			}
		}
	}

	if (_outSize > 1) {
		for (size_t i = 0; i < _outSize - 1; i++)
		{
			out_data[index + i * 2] = out_data[index + i * 2 + 2];
			out_data[index + i * 2 + 1] = (float)((out_data[index + i * 2 + 3] - out_data[index + i * 2 + 1]) * in_h);
		}
		_outSize = _outSize * 1 - 1;
	}
	else {
		_outSize = 0;
	}

	out_dataSizes[idx] = _outSize;
	return _outSize;
}

__device__ void kdeMethod(int idx,
	float* data,
	int* kdeResult,
	int kdeSampling,
	int _outSize,
	float kdeSamplesInterval1,
	float kdeSamplesInterval2,
	size_t amountOfTPoints,
	float kdeSmoothH,
	int criticalValueOfPeaks)
{
	float k1 = kdeSampling * _outSize;
	float k2 = (kdeSamplesInterval2 - kdeSamplesInterval1) / (k1 - 1);
	float delt = 0;
	float prevPrevData2 = 0;
	float prevData2 = 0;
	float data2 = 0;
	float memoryData2 = 0;
	bool strangePeak = false;
	int resultKde = 0;

	if (_outSize == 0)
	{
		kdeResult[idx] = 0;
		return;
	}

	if (_outSize == 1)
	{
		kdeResult[idx] = 1;
		return;
	}

	if (_outSize == 2)
	{
		kdeResult[idx] = 1;
		return;
	}

	if (_outSize > criticalValueOfPeaks)
	{
		kdeResult[idx] = criticalValueOfPeaks;
		return;
	}

	for (int w = 0; w < k1 - 1; ++w)
	{
		delt = w * k2 + kdeSamplesInterval1;
		prevPrevData2 = prevData2;
		prevData2 = data2;
		data2 = 0;
		for (int m = 0; m < _outSize; ++m)
		{
			double tempData = (data[idx * amountOfTPoints + m] - delt) / kdeSmoothH;
			data2 += expf(-((tempData * tempData) / 2));
		}
		// Íàéòè çäåñü - ÿâëÿåòñÿ ëè çäåñü data2 ïèêîì èëè íåò. Åñëè äà - èíêðåìèðóåì resultKde
		if (w < 2)
			continue;

		if (strangePeak)
		{
			if (prevData2 == data2)
				continue;
			else if (prevData2 < data2)
			{
				strangePeak = false;
				continue;
			}
			else if (prevData2 > data2)
			{
				strangePeak = false;
				++resultKde;
				continue;
			}
		}
		else if (prevData2 > prevPrevData2 && prevData2 > data2)
		{
			++resultKde;
			continue;
		}
		else if (prevData2 > prevPrevData2 && prevData2 == data2)
		{
			strangePeak = true;
			memoryData2 = prevData2;
			continue;
		}
	}
	if (prevData2 < data2)
	{
		++resultKde;
	}
	kdeResult[idx] = resultKde;
	return;
}




template <class T1, class T2>
__host__ void linspace(T1 a, T1 b, int amount, T2 * out, int startIndex)
{
	if (amount <= 0)
		throw std::invalid_argument("linspace error. amount <= 0");
	if (amount == 1)
	{
		out[0] = a;
		return;
	}

	double step = (b - a) / (amount - 1);
	for (size_t i = 0; i < amount; ++i)
		out[startIndex + i] = a + i * step;

	return;
}

__device__ float customAbs(float value)
{
	if (value < 0)
		return -value;
	return value;
}


__device__ float distance(float x1, float y1, float x2, float y2)
{
	if (x1 == x2 && y1 == y2)
		return 0;
	float dx = x2 - x1;
	float dy = y2 - y1;

	return hypotf(dx, dy);
}


__device__ void expand_cluster(float* input, int index, int amountOfPeaks, int p, float eps)
{
	for (int i = index + 0; i < index + (amountOfPeaks * 2 - 2); i += 2) {
		if (distance(input[i + 1], input[i + 3], input[p + 1], input[p + 3]) < eps) {
			float temp = input[i + 1];
			input[i + 1] = temp > 0 ? -temp : temp;

			if (i != p)
				expand_cluster(input, index, amountOfPeaks, i, eps);
		}
	}
}


__device__ int dbscan(float* input, int amountOfTPoints, int amountOfPeaks, int idx, float eps, int* dataSizes, int criticalValueOfPeaks)
{
	if (amountOfPeaks <= 0)
	{
		dataSizes[idx] = 0;
		return;
	}

	if (amountOfPeaks == 1)
	{
		dataSizes[idx] = 1;
		return;
	}


	if (amountOfPeaks > criticalValueOfPeaks)
	{
		dataSizes[idx] = 0;
		return;
	}


	//dbscan(in_data, amountOfTPoints, outSize, idx, 0.5f, in_dataSizes);
	int cluster = 0;
	int NumNeibor = 0;


	int index = amountOfTPoints * idx;

	for (int i = index + amountOfPeaks * 2; i < index + amountOfPeaks * 4; i++) {
		input[i] = 0;
	}

	for (int i = 0; i < amountOfPeaks; i++)
		if (NumNeibor >= 1)
		{
			i = input[index + amountOfPeaks * 3 + NumNeibor - 1];
			input[index + amountOfPeaks * 3 + NumNeibor - 1] = 0;
			NumNeibor = NumNeibor - 1;
			for (int k = 0; k < amountOfPeaks - 1; k++) {
				if (i != k && input[index + amountOfPeaks * 2 + k] == 0) {
					//if (distance(input[index + i], input[index + i + 1], input[index + k], input[index + k + 1])<= eps) {
					if (distance(input[index + i * 2], input[index + i * 2 + 1], input[index + k * 2], input[index + k * 2 + 1]) < eps) {
						input[index + amountOfPeaks * 2 + k] = cluster;
						input[index + amountOfPeaks * 3 + k] = k;
						NumNeibor++;
					}
				}
			}
		}
		else if (input[index + amountOfPeaks * 2 + i] == 0) {
			NumNeibor = 0;
			cluster++;
			input[index + amountOfPeaks * 2 + i] = cluster;
			for (int k = 0; k < amountOfPeaks - 1; k++) {
				if (i != k && input[index + amountOfPeaks * 2 + k] == 0) {
					//if (distance(input[index + i], input[index + i + 1], input[index + k], input[index + k + 1])<= eps) {
					if (distance(input[index + i * 2], input[index + i * 2 + 1], input[index + k * 2], input[index + k * 2 + 1]) < eps) {
						input[index + amountOfPeaks * 2 + k] = cluster;
						input[index + amountOfPeaks * 3 + k] = k;
						NumNeibor++;
					}
				}
			}
		}
	//for (int i = 0; i < amountOfPeaks; i++) {

	//}
	//for (int i = index + amountOfPeaks * 2; i < index + amountOfPeaks * 4; i++) {
	//	input[i] = 0;
	//}

	dataSizes[idx] = cluster;

	return;// cluster - 1;
}


__host__ void getParamsAndSymmetry2D(double* param1, double* param2,
	double startInterval1, double finishInteraval1,
	double startInterval2, double finishInteraval2,
	int nPts)
{
	double* tempParams = new double[nPts];
	linspace(startInterval2, finishInteraval2, nPts, tempParams);

	for (int i = 0; i < nPts; ++i)
	{
		linspace(startInterval1, finishInteraval1, nPts, param1, i * nPts);
		for (int j = 0; j < nPts; ++j)
			param2[nPts * i + j] = tempParams[i];
	}

	delete[] tempParams;
}


__host__ void getParamsAndSymmetry3D(double* param1, double* param2, double* param3,
	double startInterval1, double finishInteraval1,
	double startInterval2, double finishInteraval2,
	double startInterval3, double finishInteraval3,
	int nPts)
{
	{
		double* tempParams2 = new double[nPts];
		double* tempParams3 = new double[nPts];

		linspace(startInterval2, finishInteraval2, nPts, tempParams2);
		linspace(startInterval3, finishInteraval3, nPts, tempParams3);

		for (int k = 0; k < nPts; ++k)
			for (int i = 0; i < nPts; ++i)
			{
				linspace(startInterval1, finishInteraval1, nPts, param1, i * nPts + k * nPts * nPts);
				for (int j = 0; j < nPts; ++j)
				{
					param2[nPts * nPts * k + nPts * i + j] = tempParams2[i];
					param3[nPts * nPts * k + nPts * i + j] = tempParams3[k];
				}
			}

		delete[] tempParams2;
		delete[] tempParams3;
	}
}


template <class T>
__host__ void slice(T * in, int a, int b, T * out)
{
	if (b - a < 0)
		throw std::invalid_argument("slice error. b < a");
	for (size_t i = 0; i < b - a; ++i)
		out[i] = in[a + i];
}


